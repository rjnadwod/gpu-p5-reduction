
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

__global__ void reduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE
    __shared__ float sdata[2 * BLOCK_SIZE];
    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * BLOCK_SIZE;

    if (start + t < size)
       sdata[t] = in[start + t];
    else
       sdata[t] = 0;

    if (start + BLOCK_SIZE + t < size)
       sdata[BLOCK_SIZE + t] = in[start + BLOCK_SIZE + t];
    else
       sdata[BLOCK_SIZE + t] = 0;

    for (unsigned int stride = BLOCK_SIZE; stride >= 1; stride >>= 1) {
       __syncthreads();
       if (t < stride)
          sdata[t] += sdata[t+stride];
    }

    if (t == 0)
       out[blockIdx.x] = sdata[0];
}
